#include "hip/hip_runtime.h"
/*
  ======================================================================
   demo.c --- protoype to show off the simple solver
  ----------------------------------------------------------------------
   Author : Jos Stam (jstam@aw.sgi.com)
   Creation Date : Jan 9 2003

   Description:

        This code is a simple prototype that demonstrates how to use the
        code provided in my GDC2003 paper entitles "Real-Time Fluid Dynamics
        for Games". This code uses OpenGL and GLUT for graphics and interface

  =======================================================================
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#include "indices.h"
#include "solver.cuh"
#include "timing.h"


/* macros */

#define IX(x, y) (rb_idx((x), (y), (N + 2)))


/* global variables */

static int N;
static float dt, diff, visc;
static float force, source;

static float *u, *v, *u_prev, *v_prev;
static float *dens, *dens_prev;


/*
  ----------------------------------------------------------------------
   free/clear/allocate simulation data
  ----------------------------------------------------------------------
*/


static void free_data(void)
{
    if (u) {
        hipFree(u);
    }
    if (v) {
        hipFree(v);
    }
    if (u_prev) {
        hipFree(u_prev);
    }
    if (v_prev) {
        hipFree(v_prev);
    }
    if (dens) {
        hipFree(dens);
    }
    if (dens_prev) {
        hipFree(dens_prev);
    }
}

static void clear_data(void)
{
    int size = (N + 2) * (N + 2);
    hipMemset(u, 0, size * sizeof(float));
    hipMemset(v, 0, size * sizeof(float));
    hipMemset(u_prev, 0, size * sizeof(float));
    hipMemset(v_prev, 0, size * sizeof(float));
    hipMemset(dens, 0, size * sizeof(float));
    hipMemset(dens_prev, 0, size * sizeof(float));
}

static int allocate_data(void)
{
    int size = (N + 2) * (N + 2);

    hipMalloc(&u, size * sizeof(float));
    hipMalloc(&v, size * sizeof(float));
    hipMalloc(&u_prev, size * sizeof(float));
    hipMalloc(&v_prev, size * sizeof(float));
    hipMalloc(&dens, size * sizeof(float));
    hipMalloc(&dens_prev, size * sizeof(float));

    if (!u || !v || !u_prev || !v_prev || !dens || !dens_prev) {
        fprintf(stderr, "cannot allocate data\n");
        return (0);
    }

    return (1);
}

// Usar lo enseñado en cuda 4 aca
__global__ void compute_max_kernel(const float* d, const float* u, const float* v, int size, float* max_velocity2, float* max_density)
{
    extern __shared__ float sdata[];
    float* smax_vel2 = sdata;
    float* smax_dens = sdata + blockDim.x;

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    float local_vel2 = 0.0f;
    float local_dens = 0.0f;

    if (i < size) {
        local_vel2 = u[i] * u[i] + v[i] * v[i];
        local_dens = d[i];
    }

    smax_vel2[tid] = local_vel2;
    smax_dens[tid] = local_dens;
    __syncthreads();

    // Parallel reduction for max
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (smax_vel2[tid] < smax_vel2[tid + s])
                smax_vel2[tid] = smax_vel2[tid + s];
            if (smax_dens[tid] < smax_dens[tid + s])
                smax_dens[tid] = smax_dens[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        max_velocity2[blockIdx.x] = smax_vel2[0];
        max_density[blockIdx.x] = smax_dens[0];
    }
}

// Device kernel to clear arrays
__global__ void clear_arrays_kernel(float* d, float* u, float* v, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        d[i] = 0.0f;
        u[i] = 0.0f;
        v[i] = 0.0f;
    }
}

// Device kernel to set a value at a specific index
__global__ void set_value_kernel(float* arr, int idx, float value)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        arr[idx] = value;
    }
}

static void react(float* d, float* u, float* v)
{
    int size = (N + 2) * (N + 2);

    // Compute max velocity^2 and max density on device
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    float* d_max_velocity2;
    float* d_max_density;
    hipMalloc(&d_max_velocity2, blocks * sizeof(float));
    hipMalloc(&d_max_density, blocks * sizeof(float));

    compute_max_kernel<<<blocks, threads, threads * 2 * sizeof(float)>>>(d, u, v, size, d_max_velocity2, d_max_density);

    // Reduce on host
    float* h_max_velocity2 = (float*)malloc(blocks * sizeof(float));
    float* h_max_density = (float*)malloc(blocks * sizeof(float));
    hipMemcpy(h_max_velocity2, d_max_velocity2, blocks * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_max_density, d_max_density, blocks * sizeof(float), hipMemcpyDeviceToHost);

    float max_velocity2 = 0.0f;
    float max_density = 0.0f;
    for (int i = 0; i < blocks; i++) {
        if (max_velocity2 < h_max_velocity2[i])
            max_velocity2 = h_max_velocity2[i];
        if (max_density < h_max_density[i])
            max_density = h_max_density[i];
    }

    free(h_max_velocity2);
    free(h_max_density);
    hipFree(d_max_velocity2);
    hipFree(d_max_density);

    // Clear arrays on device
    clear_arrays_kernel<<<blocks, threads>>>(d, u, v, size);

    // Set initial values if needed
    if (max_velocity2 < 0.0000005f) {
        int idx = IX(N / 2, N / 2);
        set_value_kernel<<<1, 1>>>(u, idx, force * 10.0f);
        set_value_kernel<<<1, 1>>>(v, idx, force * 10.0f);
    }
    if (max_density < 1.0f) {
        int idx = IX(N / 2, N / 2);
        set_value_kernel<<<1, 1>>>(d, idx, source * 10.0f);
    }
}
static void one_step(void)
{
    static int times = 1;
    static double start_t = 0.0;
    static double one_second = 0.0;
    static double react_ns_p_cell = 0.0;
    static double vel_ns_p_cell = 0.0;
    static double dens_ns_p_cell = 0.0;

    start_t = wtime();
    react(dens_prev, u_prev, v_prev);
    react_ns_p_cell += 1.0e9 * (wtime() - start_t) / (N * N);

    start_t = wtime();
    vel_step(N, u, v, u_prev, v_prev, visc, dt);
    vel_ns_p_cell += 1.0e9 * (wtime() - start_t) / (N * N);

    start_t = wtime();
    dens_step(N, dens, dens_prev, u, v, diff, dt);
    dens_ns_p_cell += 1.0e9 * (wtime() - start_t) / (N * N);

    if (1.0 < wtime() - one_second) { /* at least 1s between stats */
        printf("%lf, %lf, %lf, %lf\n",
               (react_ns_p_cell + vel_ns_p_cell + dens_ns_p_cell) / times,
               react_ns_p_cell / times, vel_ns_p_cell / times, dens_ns_p_cell / times);
        one_second = wtime();
        react_ns_p_cell = 0.0;
        vel_ns_p_cell = 0.0;
        dens_ns_p_cell = 0.0;
        times = 1;
    } else {
        times++;
    }
}


/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main(int argc, char** argv)
{
    int i = 0;

    if (argc != 1 && argc != 2 && argc != 7) {
        fprintf(stderr, "usage : %s N dt diff visc force source\n", argv[0]);
        fprintf(stderr, "where:\n");
        fprintf(stderr, "\t N      : grid resolution\n");
        fprintf(stderr, "\t dt     : time step\n");
        fprintf(stderr, "\t diff   : diffusion rate of the density\n");
        fprintf(stderr, "\t visc   : viscosity of the fluid\n");
        fprintf(stderr, "\t force  : scales the mouse movement that generate a force\n");
        fprintf(stderr, "\t source : amount of density that will be deposited\n");
        exit(1);
    }

    if (argc == 1 || argc == 2) {
        N = (argc == 1) ? 128 : atoi(argv[1]);
        dt = 0.1f;
        diff = 0.0f;
        visc = 0.0f;
        force = 5.0f;
        source = 100.0f;
        fprintf(stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
                N, dt, diff, visc, force, source);
    } else {
        N = atoi(argv[1]);
        dt = atof(argv[2]);
        diff = atof(argv[3]);
        visc = atof(argv[4]);
        force = atof(argv[5]);
        source = atof(argv[6]);
    }

    if (!allocate_data()) {
        exit(1);
    }
    clear_data();
    for (i = 0; i < 2048; i++) {
        one_step();
    }
    free_data();

    exit(0);
}
